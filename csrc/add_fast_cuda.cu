#include "hip/hip_runtime.h"

#include <iostream> // for debugging
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

// // TODO inline func instead of macro
// #define AS_DENSE_ND(TENSOR, NDIMS) ((TENSOR).packed_accessor32<scalar_t, NDIMS, torch::RestrictPtrTraits>())
// #define AS_DENSE_1D(TENSOR) AS_DENSE_ND(TENSOR, 1)
// #define AS_DENSE_2D(TENSOR) AS_DENSE_ND(TENSOR, 2)
// #define AS_DENSE_3D(TENSOR) AS_DENSE_ND(TENSOR, 3)


namespace {

// template<typename T> using Dense1d = torch::PackedTensorAccessor32<T, 1, torch::RestrictPtrTraits>;

// template <int bytes_per_elem> struct byte_count_traits {};

// template<> struct byte_count_traits<1> { using dtype = uint8_t; };
// template<> struct byte_count_traits<2> { using dtype = uint16_t; };
// template<> struct byte_count_traits<4> { using dtype = uint32_t; };
// template<> struct byte_count_traits<8> { using dtype = uint64_t; };
// template<> struct byte_count_traits<16> { using dtype = double2; }; // cuda vec dtype

// template<typename scalar_t, int bytes_per_thread>
// __global__ void _add_fast_kernel(const Dense1d<scalar_t> a,
//                                  const Dense1d<scalar_t> b,
//                                  Dense1d<scalar_t> c,
//                                  size_t N) {
//     using load_as_dtype = typename byte_count_traits<bytes_per_thread>::dtype;
//     static constexpr auto elem_sz = sizeof(a[0]);
//     static_assert(bytes_per_thread >= elem_sz);
//     static constexpr auto elems_per_read = bytes_per_thread / elem_sz;

//     auto a_ptr = reinterpret_cast<const load_as_dtype*>(&a[0]);
//     auto b_ptr = reinterpret_cast<const load_as_dtype*>(&b[0]);
//     auto c_ptr = reinterpret_cast<load_as_dtype*>(&c[0]);

//     int total_vec_reads = N * elem_sz / bytes_per_thread;
//     int num_non_stragglers = total_vec_reads * elems_per_read;
//     int num_stragglers = N - num_non_stragglers;
//     int index_in_grid = blockIdx.x * blockDim.x + threadIdx.x;

//     // grid stride loop to allow grid size smaller than numel; see:
//     // https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
//     int grid_numel = blockDim.x * gridDim.x;
//     for (int i = index_in_grid; i < total_vec_reads; i += grid_numel) {
//         // load data as if load_as_dtype is the true dtype. Then operate
//         // on it as the original dtype. We use the volatile keyword to
//         // hopefully make the compiler issue loads of the wider dtype
//         // instead of just flattening the pointer arithmetic and doing
//         // loads as the true dtype.
//         // const volatile load_as_dtype a_vec = a_ptr[i];
//         // const volatile load_as_dtype b_vec = b_ptr[i];
//         // volatile load_as_dtype c_vec = c_ptr[i];
//         const load_as_dtype a_vec = __ldg(&a_ptr[i]);
//         const load_as_dtype b_vec = __ldg(&b_ptr[i]);
//         load_as_dtype c_vec = __ldg(&c_ptr[i]);

//         // version that runs correctly, but super slowly
//         // auto a_vec_ptr = reinterpret_cast<const scalar_t*>(&a_vec);
//         // auto b_vec_ptr = reinterpret_cast<const scalar_t*>(&b_vec);
//         // auto c_vec_ptr = reinterpret_cast<scalar_t*>(&c_vec);
//         // for (int ii = 0; ii < elems_per_read; ii++) {
//         //     c_vec_ptr[ii] = a_vec_ptr[ii] + b_vec_ptr[ii];
//         // }
//         // // c_ptr[i] = *reinterpret_cast<load_as_dtype*>(&c_vec);  // cast to rm volatile
//         // c_ptr[i] = c_vec;

//         // version that hopefully runs fast?
//         scalar_t a_ar[elems_per_read];
//         scalar_t b_ar[elems_per_read];
//         scalar_t c_ar[elems_per_read];
//         std::memcpy(&a_ar, &a_vec, sizeof(load_as_dtype));
//         std::memcpy(&b_ar, &b_vec, sizeof(load_as_dtype));
//         std::memcpy(&c_ar, &c_vec, sizeof(load_as_dtype));

//         // auto a_vec_ar = reinterpret_cast<scalar_t[elems_per_read]>(&a_vec);
//         // auto b_vec_ar = reinterpret_cast<scalar_t[elems_per_read]>(&b_vec);
//         // auto c_vec_ar = reinterpret_cast<scalar_t[elems_per_read]>(&c_vec);
//         for (int ii = 0; ii < elems_per_read; ii++) {
//             c_ar[ii] = a_ar[ii] + b_ar[ii];
//         }
//         // c_ptr[i] = *reinterpret_cast<load_as_dtype*>(&c_vec);  // cast to rm
//         // volatile
//         std::memcpy(&c_ptr[i], &c_ar, sizeof(load_as_dtype));
//         // c_ptr[i] = c_vec_ar;
//     }
//     // handle trailing elems, if any, using scalar loads in the true dtype
//     if (index_in_grid < num_stragglers) {
//         auto idx = num_non_stragglers + index_in_grid;
//         c[idx] = a[idx] + b[idx];
//     }
// }

__global__ void _add_fast_f32(const float* __restrict__ a_tensor,
                              const float* __restrict__ b_tensor,
                              float* __restrict__ c_tensor,
                              uint32_t N,
                              uint32_t total_vec_reads)
{
    using load_as_dtype = float4;
    static constexpr uint32_t elem_sz = sizeof(float);
    static constexpr uint32_t bytes_per_thread = sizeof(load_as_dtype);
    static constexpr uint32_t elems_per_read = bytes_per_thread / elem_sz;
    static_assert(elems_per_read == 1 || elems_per_read == 2 ||
                  elems_per_read == 4);

    auto a_ptr = reinterpret_cast<const load_as_dtype*>(&a_tensor[0]);
    auto b_ptr = reinterpret_cast<const load_as_dtype*>(&b_tensor[0]);
    auto c_ptr = reinterpret_cast<load_as_dtype*>(&c_tensor[0]);

    // auto total_vec_reads = N / elems_per_read;
    auto num_non_stragglers = total_vec_reads * elems_per_read;
    auto num_stragglers = N - num_non_stragglers;
    auto index_in_grid = blockIdx.x * blockDim.x + threadIdx.x;

    // grid stride loop to allow grid size smaller than numel; see:
    // https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    auto grid_numel = blockDim.x * gridDim.x;
    for (uint32_t i = index_in_grid; i < total_vec_reads; i += grid_numel) {
        const load_as_dtype a = __ldg(&a_ptr[i]);
        const load_as_dtype b = __ldg(&b_ptr[i]);
        load_as_dtype c = c_ptr[i];

        c.x = a.x + b.x;
        c.y = a.y + b.y;
        c.z = a.z + b.z;
        c.w = a.w + b.w;
        c_ptr[i] = c;
    }
    if (index_in_grid < num_stragglers) {
        auto idx = num_non_stragglers + index_in_grid;
        c_tensor[idx] = a_tensor[idx] + b_tensor[idx];
    }
}

// __global__ void _add_fast_f32(const Dense1d<float> a_tensor,
//                               const Dense1d<float> b_tensor,
//                               Dense1d<float> c_tensor, size_t N) {
//     using load_as_dtype = float4;
//     static constexpr size_t elem_sz = sizeof(float);
//     static constexpr size_t bytes_per_thread = sizeof(load_as_dtype);
//     static constexpr size_t elems_per_read = bytes_per_thread / elem_sz;
//     static_assert(elems_per_read == 1 || elems_per_read == 2 ||
//                   elems_per_read == 4);

//     auto a_ptr = reinterpret_cast<const load_as_dtype*>(&a_tensor[0]);
//     auto b_ptr = reinterpret_cast<const load_as_dtype*>(&b_tensor[0]);
//     auto c_ptr = reinterpret_cast<load_as_dtype*>(&c_tensor[0]);

//     auto total_vec_reads = N / elems_per_read;
//     auto num_non_stragglers = total_vec_reads * elems_per_read;
//     auto num_stragglers = N - num_non_stragglers;
//     auto index_in_grid = blockIdx.x * blockDim.x + threadIdx.x;

//     // grid stride loop to allow grid size smaller than numel; see:
//     // https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
//     auto grid_numel = blockDim.x * gridDim.x;
//     for (int i = index_in_grid; i < total_vec_reads; i += grid_numel) {
//         const load_as_dtype a = __ldg(&a_ptr[i]);
//         const load_as_dtype b = __ldg(&b_ptr[i]);
//         load_as_dtype c = c_ptr[i];

//         c.x = a.x + b.x;
//         c.y = a.y + b.y;
//         c.z = a.z + b.z;
//         c.w = a.w + b.w;
//         c_ptr[i] = c;
//         // switch (elems_per_read) {
//         //     case 1:
//         //         c.x = a.x + b.x;
//         //     case 2:
//         //         c.y = a.y + b.y;
//         //     case 4:
//         //         // makes it not compile with float2 loads since
//         //         // `error: class "float2" has no member "z"`
//         //         c.z = a.z + b.z;
//         //         c.w = a.x + b.w;
//         // }
//     }
//     if (index_in_grid < num_stragglers) {
//         auto idx = num_non_stragglers + index_in_grid;
//         c_tensor[idx] = a_tensor[idx] + b_tensor[idx];
//     }
// }

}  // anon namespace

// template<typename scalar_t>
// void _add_fast_static_bytes_per_kernel(const Dense1d<scalar_t> a,
//                                        const Dense1d<scalar_t> b,
//                                        Dense1d<scalar_t> c,
//                                        size_t N,
//                                        size_t bytes_per_thread,
//                                        size_t grid_size,
//                                        size_t block_size) {
//     bytes_per_thread = max(bytes_per_thread, sizeof(scalar_t));
//     // assert(bytes_per_thread >= sizeof(scalar_t));
//     // auto bytes_per_block = bytes_per_thread * block_shape;
//     // auto min_bytes = grid_size * bytes_per_block;
//     // auto min_elems = min_bytes / sizeof(scalar_t);
//     // assert(block_size % 32 * sizeof(scalar_t) / bytes_per_thread = )

//     dim3 grid_shape = grid_size;
//     dim3 block_shape = block_size;
//     switch (bytes_per_thread) {
//     // case 1:
//     //     _add_fast_kernel<scalar_t, 1><<<grid_shape, block_shape>>>(a, b, c, N);
//     //     break;
//     // case 2:
//     //     _add_fast_kernel<scalar_t, 2><<<grid_shape, block_shape>>>(a, b, c, N);
//     //     break;
//     case 4:
//         _add_fast_kernel<scalar_t, 4><<<grid_shape, block_shape>>>(a, b, c, N);
//         break;
//     case 8:
//         _add_fast_kernel<scalar_t, 8><<<grid_shape, block_shape>>>(a, b, c, N);
//         break;
//     case 16:
//         _add_fast_kernel<scalar_t, 16><<<grid_shape, block_shape>>>(a, b, c, N);
//         break;
//     }
// }

inline size_t div_round_up(size_t x, size_t y) {
    return (x + y - 1) / y;
}

inline int num_sms() {
    int deviceID;
    hipDeviceProp_t props;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&props, deviceID);
    return props.multiProcessorCount;
}

// note that this requires you to do a grid-stride loop, since it doesn't
// guarantee a thread count proportional to the input size N
inline int grid_size_for_block_size(size_t block_size, size_t N, size_t numel_per_thread=1) {
    constexpr size_t kMaxBlocksPerSM = 32;
    auto numel = N / numel_per_thread;
    auto sm_count = num_sms();
    // auto max_threads = sm_count * 2048;  // highest occupancy possible
    auto max_simultaneous_blocks = sm_count * kMaxBlocksPerSM;
    auto max_blocks_of_work = div_round_up(numel, block_size);
    return min(max_simultaneous_blocks, max_blocks_of_work);
    // auto max_grid_size = div_round_up(max_threads, block_size);
    // return min(max_blocks_of_work, max_grid_size);
}

void add_fast_wrapper(const at::Tensor in_a, const at::Tensor in_b,
                      at::Tensor out_c, size_t block_size, int grid_size)
{
    constexpr int numel_per_thread = 4;  // 4 for fast version
    auto N = in_a.numel();
    int num_blocks = grid_size;
    if (grid_size <= 0) {
        auto num_blocks = grid_size_for_block_size(block_size, N, numel_per_thread);
        // num_blocks = div_round_up(N, block_size * numel_per_thread); // TODO is this faster?
    }

    // std::cout << "num blocks: " << num_blocks;
    // std::cout << " block size: " << block_size << std::endl;

    // auto sm_count = num_cuda_cores();
    // size_t max_threads = sm_count * 2048;  // highest occupancy possible

    // // TODO use bytes_per_thread

    // // block_size = min(32 * (N / 32), block_size);
    // size_t num_blocks = div_round_up(N, block_size);
    // num_blocks = min(num_blocks, max_threads / block_size);
    // // dim3 grid_shape = num_blocks;

    // for available dispatch macro options, see here:
    // https:// github.com/pytorch/pytorch/blob/a2988c9e6ac281c2bf88eefde7fdd8ead44a8b36/aten/src/ATen/Dispatch.h
    // you'd think AT_DISPATCH_ALL_TYPES would include fp16 and bf16, but it
    // doesn't. Also, this macro still doesn't include bool, although
    // that's probably for the best.
    // AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
    // AT_DISPATCH_FLOATING_TYPES(
    //     in_a.type(), "add_fast_cuda", ([&] {
    //         _add_fast_static_bytes_per_kernel<scalar_t>(
    //             AS_DENSE_1D(in_a), AS_DENSE_1D(in_b), AS_DENSE_1D(out_c),
    //             N, bytes_per_thread, num_blocks, block_size);
    // }));

    // this is the more manual version that lets you choose specific dtypes
    // note that the function call has to be wrapped in a lambda for the macro
    // to work; these macros aren't needed for the op to run; they just add
    // some logging + error handling, defines scalar_t, and maps torch
    // dtypes to cpp types.
        // in_a.type(), "add_fast_cuda",
    // AT_DISPATCH_CASE(at::ScalarType::Float, [&] {
    // AT_DISPATCH_SWITCH(in_a.type(), "add_fast_cuda",
    //     AT_DISPATCH_CASE_FLOATING_TYPES([&] {
    //         _add_fast_static_bytes_per_kernel<scalar_t>(
    //             AS_DENSE_1D(in_a), AS_DENSE_1D(in_b),
    //             AS_DENSE_1D(out_c), N, bytes_per_thread, num_blocks,
    //             block_size);
    //         };
    //     )
    // );
    const auto& the_type = in_a.type();
    switch (in_a.scalar_type()) {
    case at::ScalarType::Float:
        // using scalar_t = float; // for AS_DENSE macros
        // at::ScalarType _st = ::detail::scalar_type(the_type);
        auto total_vec_reads = N / numel_per_thread;
        _add_fast_f32<<<num_blocks, block_size>>>(
            in_a.data_ptr<float>(),
            in_b.data_ptr<float>(),
            out_c.data_ptr<float>(),
            N, total_vec_reads);
        // _add_fast_static_bytes_per_kernel<scalar_t>(
        //     AS_DENSE_1D(in_a), AS_DENSE_1D(in_b), AS_DENSE_1D(out_c), N,
        //     bytes_per_thread, num_blocks, block_size);
        break;
    }

// #define AT_DISPATCH_SWITCH(TYPE, NAME, ...)                                  \
//     [&] {                                                                    \
//         const auto& the_type = TYPE;                                         \
//         constexpr const char* at_dispatch_name = NAME;                       \
//         /* don't use TYPE again in case it is an expensive or side-effect op \
//          */                                                                  \
//         at::ScalarType _st = ::detail::scalar_type(the_type);                \
//         RECORD_KERNEL_FUNCTION_DTYPE(at_dispatch_name, _st);                 \
//         switch (_st) {                                                       \
//             __VA_ARGS__                                                      \
//             default:                                                         \
//                 AT_ERROR('"', at_dispatch_name, "\" not implemented for '",  \
//                          toString(_st), "'");                                \
//         }                                                                    \
//     }()
}
