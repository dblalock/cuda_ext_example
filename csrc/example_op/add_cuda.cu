#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// TODO inline func instead of macro
#define AS_DENSE_ND(TENSOR, NDIMS) (TENSOR).packed_accessor<scalar_t, NDIMS, torch::RestrictPtrTraits, size_t>()
#define AS_DENSE_1D(TENSOR) AS_DENSE_ND(TENSOR, 1)
#define AS_DENSE_2D(TENSOR) AS_DENSE_ND(TENSOR, 2)
#define AS_DENSE_3D(TENSOR) AS_DENSE_ND(TENSOR, 3)

template<typename T> using Dense1d = torch::PackedTensorAccessor<T, 1, torch::RestrictPtrTraits, size_t>;

namespace {

template <typename scalar_t>
__global__ void _add_kernel(const Dense1d<scalar_t> a,
                            const Dense1d<scalar_t> b,
                            Dense1d<scalar_t> c,
                            size_t N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}

} // anon namespace

void add_wrapper(at::Tensor in_a, at::Tensor in_b, at::Tensor out_c, int block_size = 64) {
    size_t N = in_a.numel();
    dim3 grid_shape = N / block_size;
    // for available dispatch macro options, see here:
    // https:// github.com/pytorch/pytorch/blob/a2988c9e6ac281c2bf88eefde7fdd8ead44a8b36/aten/src/ATen/Dispatch.h
    // you'd think AT_DISPATCH_ALL_TYPES would include fp16 and bf16, but it
    // doesn't. Also, this macro still doesn't include bool, although
    // that's probably for the best.
    // AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
    AT_DISPATCH_FLOATING_TYPES(in_a.type(), "add_cuda", ([&] {
            _add_kernel<scalar_t><<<grid_shape, block_size>>>(
                AS_DENSE_1D(in_a), AS_DENSE_1D(in_b), AS_DENSE_1D(out_c), N);
    }));
}
